#include "hip/hip_runtime.h"
#include <builtin_types.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>
#include "../drvapi_error_string.h"

using namespace std;
	__global__
void saxpy(int n, float a, float *x, float *y)
{
	for (int i=0 ; i<10000000000 ; i++)
	{
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i < n) y[i] = a*x[i] + y[i];
		 asm("exit;");
	}
}

void prepareSaxpy()
{

	int N = 1<<20;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));

	hipMalloc(&d_x, N*sizeof(float)); 
	hipMalloc(&d_y, N*sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

	// Perform SAXPY on 1M elements
	saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

	hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

}

int main(int argc, char *argv[])
{
	struct timeval popContext_1, popContext_2, cudaCallStart_1, cudaCallEnd_1;
	struct timeval cudaCallStart_2, cudaCallEnd_2, cudaCallStart_3, cudaCallEnd_3;
	hipCtx_t cuCurrent = NULL;
	hipError_t result;
	hipDevice_t cuDevice; 
	char name[100];
	
	result = hipInit(0);
	if (result != hipSuccess)
	{
		printf("hipDeviceGet: %d\n", getCudaDrvErrorString(result));
		return 0;
	}

	//Get device of the current thread
	//result = hipCtxGetDevice (&cuDevice);
	result = hipDeviceGet (&cuDevice, 1);
	if (result != hipSuccess)
	{
		printf("hipDeviceGet: %d\n", getCudaDrvErrorString(result));
		return 0;
	}


	result = hipCtxCreate(&cuCurrent, 0, cuDevice);
	if (result != hipSuccess)
	{
		printf("hipCtxCreate: %d\n", getCudaDrvErrorString(result));
		return 0;
	}

	hipDeviceGetName(name, 100, cuDevice);
	cout<<"Name "<<name<<"  of dev: "<<cuDevice<<endl;

	/*   -- Get context of a thread -- */
	gettimeofday(&popContext_1,NULL);
	double t1 = popContext_1.tv_sec  * 1000000 +  popContext_1.tv_usec;
	{
		//Get context of the current thread
		result = hipCtxPopCurrent(&cuCurrent);
		if (result != hipSuccess)
		{
			printf("hipCtxPopCurrent: %d\n", result);
			return 0;
		}
	}
	gettimeofday(&popContext_2,NULL);
	double t2 = popContext_2.tv_sec  * 1000000 +  popContext_2.tv_usec;                                
	long double durationofctx = (t2 - t1)/1000 ;
	cout<<"Duration of getContext: "<<durationofctx<<endl;

	/*   -- DONE: Get context of a thread -- */
	//prepareSaxpy();
	/*   -- Destroy context of a thread -- */
	gettimeofday(&cudaCallStart_1,NULL);
	double cuda_t1 = cudaCallStart_1.tv_sec  * 1000000 +  cudaCallStart_1.tv_usec;
	{
		result = hipCtxDestroy(cuCurrent);
		if (result != hipSuccess)
		{
			printf("hipCtxDestroy: %d\n", result);
			return 0;
		}

	}
	gettimeofday(&cudaCallEnd_1,NULL);
	double cuda_t2 =  cudaCallEnd_1.tv_sec  * 1000000 +  cudaCallEnd_1.tv_usec;
	long double durationCudaCall = (cuda_t2 -cuda_t1)/1000 ;
	cout<<"Duration of destroy context: "<<durationCudaCall<<endl;
	/*   -- DONE: Destroy context of a thread -- */


	float *d_a;
	int size = sizeof(int);
	hipError_t error;

	/*   -- Create context of a thread -- */
	/*
	gettimeofday(&cudaCallStart_2,NULL);
	double cuda_t3 = cudaCallStart_2.tv_sec  * 1000000 +  cudaCallStart_2.tv_usec;
	{
		result = hipCtxCreate(&cuCurrent, 0, cuDevice);
		if (result != hipSuccess)
		{
			printf("cuCtxPopCreate: %d\n", result);
			return 0;
		}

	}
	gettimeofday(&cudaCallEnd_2,NULL);
	double cuda_t4 =  cudaCallEnd_2.tv_sec  * 1000000 +  cudaCallEnd_2.tv_usec;
	long double durationCudaCall2 = (cuda_t4 -cuda_t3)/1000 ;
	cout<<"Duration of create context: "<<durationCudaCall2<<endl;
*/
	gettimeofday(&cudaCallStart_3,NULL);
	double cuda_t5 = cudaCallStart_3.tv_sec  * 1000000 +  cudaCallStart_3.tv_usec;

	error = hipMalloc((void**)&d_a,size);

	if (error != hipSuccess)
	{
		cerr <<"Malloc failed"<<endl;
		return false;
	}
	gettimeofday(&cudaCallEnd_3,NULL);
	double cuda_t6 =  cudaCallEnd_3.tv_sec  * 1000000 +  cudaCallEnd_3.tv_usec;
	long double durationCudaCall3 = (cuda_t6 -cuda_t5)/1000 ;
	cout<<"Duration of 1st malloc: "<<durationCudaCall3<<endl;

	/*   -- DONE: Create context of a thread -- */
	return 1;
}
